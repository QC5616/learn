#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<sys/time.h>
#include<hip/hip_runtime.h>

typedef unsigned int uint32_t;
typedef unsigned long long uint64_t;

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

// logic functions
#define ROTL(W, n) (((W << n) & 0xFFFFFFFF) | (W) >> (32 - (n)))
#define SHR(W, n) ((W >> n) & 0xFFFFFFFF)
#define Conditional(x, y, z) ((x&y)^((~x)&z))
#define Majority(x, y, z) ((x&y)^(x&z)^(y&z))
#define LSigma_0(x) (ROTL(x,30)^ROTL(x,19)^ROTL(x,10))
#define LSigma_1(x) (ROTL(x,26)^ROTL(x,21)^ROTL(x,7))
#define SSigma_0(x) (ROTL(x,25)^ROTL(x,14)^SHR(x,3))
#define SSigma_1(x) (ROTL(x,15)^ROTL(x,13)^SHR(x,10))

// the path of file
const char *FILEPATH = "/home/chenq/cuda/0.txt";

// the numbers of characters per reading file 600LLU * 1024 * 1024
uint64_t READSIZE = 600LLU * 1024 * 1024;

// the size of a data block per layer
uint64_t DATABLOCKSIZE[2] = {1LL*1024*1024, 0LLU};

// the number of characters for padding per layer
uint64_t PADDINGSIZE[2] = {0LLU, 0LLU};

// 0. recording time in seconds
double getTime();

// 1. preprocess
void preprocess(const uint64_t readCharacters, uint64_t * dataBlockAmountPerReading, uint64_t *storageSizePerReading);

// 2. padding characters
__global__ void paddingChar(unsigned char* D_C, unsigned char* D_P, uint64_t DATABLOCKSIZE0, uint64_t DATABLOCKSIZE1, uint64_t PADDINGSIZE0, uint64_t PADDINGSIZE1, uint64_t dataBlockAmount);

// 3. transform 4 unsigned char to 1 32-bit unsigned int
__global__ void unsignedCharToUnsignedInt(const unsigned char* D_P, uint32_t* D_T, uint64_t DATABLOCKSIZE0, uint64_t DATABLOCKSIZE1, uint64_t PADDINGSIZE0, uint64_t PADDINGSIZE1, uint64_t dataBlockAmount);

// 4. extending 16 32-bit integers to 64 32-bit integers
__global__ void extending(uint32_t *D_T, uint32_t *D_E, uint64_t DATABLOCKSIZE0, uint64_t DATABLOCKSIZE1, uint64_t PADDINGSIZE0, uint64_t PADDINGSIZE1, uint64_t dataBlockAmount);

// 5. updating hash value
__global__ void updatingHashValue(const uint32_t *D_E, uint32_t *D_H, uint64_t DATABLOCKSIZE0, uint64_t DATABLOCKSIZE1, uint64_t PADDINGSIZE0, uint64_t PADDINGSIZE1, bool oddDataBlockAmount, uint64_t dataBlockAmount ,uint64_t hashValuePosition);

// main function
int main(int agrc, char *argv[]) {
    
    printf("\nComputing hash value on GPU.\n");

    // determining data block size
    printf("Please enter DataBlock size in Bytes: ");
    scanf("%llu", &DATABLOCKSIZE[0]);

    // set the start time
    double start, end;
    start = getTime();

    // get the file size
    printf("have read file: %s\n", argv[1]);
    FILE *fin;
    fin = fopen(argv[1], "rb");
    if (!fin) {
        printf("reading file failed.\n");
        if (agrc == 1) printf("please enter file name.\n");
        exit(EXIT_FAILURE);
    }
    fseek(fin, 0, SEEK_END);
    uint64_t fileSize = ftell(fin);
    rewind(fin);
    printf("the size of file: %llu Bytes\n", fileSize);

    // get the reading times
    if (fileSize < READSIZE) READSIZE = fileSize;
    uint64_t readTimes = fileSize / READSIZE;
    if (fileSize % READSIZE > 0) readTimes++;

    // get the number of layers in the Merkle Hash Tree
    uint64_t layers = 1;
    uint64_t layerProcess = fileSize / DATABLOCKSIZE[0];
    if (fileSize % DATABLOCKSIZE[0] > 0) layerProcess++;
    while (layerProcess != 1) {
        if (layerProcess % 2 != 0) layerProcess++;
        layerProcess = layerProcess / 2;
        layers++;
    }

    // computing hash value for 0 layer

    // 3. get the number of data block 
    uint64_t dataBlockAmount = fileSize / DATABLOCKSIZE[0];
    if (fileSize % DATABLOCKSIZE[0] > 0) dataBlockAmount++;

    // 4. determining the parity of data block amount
    bool oddDataBlockAmount = false;
    if (dataBlockAmount % 2 != 0) oddDataBlockAmount = true;

    // 5. get the number of hash value
    uint64_t hashValueAmount = dataBlockAmount;
    if ((hashValueAmount % 2 != 0) && layers > 1) hashValueAmount++;
    uint64_t hashValueAmountArray[layers];
    hashValueAmountArray[0] = hashValueAmount;

    // data stream
    char *C = NULL;
    char *D_C = NULL;

    // storing the data after padding
    unsigned char *D_P = NULL;

    //  storing the data after transform
    uint32_t *D_T = NULL;

    // storing the data after extending
    uint32_t *D_E = NULL;

    // assign the storage space of hash value
    uint32_t *D_V[layers];
    CHECK(hipMalloc((uint32_t **)&D_V[0], hashValueAmountArray[0] * 8 * sizeof(uint32_t)));

    // get data block size, padding characters, data block amount (per reading) and storage size (per reading)
    uint64_t readCharacters = READSIZE;
    if (fileSize > READSIZE && fileSize - READSIZE < 100 * 1048576) readCharacters = fileSize;
    uint64_t dataBlockAmountPerReading = 0;
    uint64_t storageSizePerReading = 0;
    preprocess(readCharacters, &dataBlockAmountPerReading, &storageSizePerReading);

    // hash value position using in computation of 0 layer
    uint64_t hashValuePosition = 0;

    // parallelly updating data block's hash value
    for (uint64_t i = 0; i < readTimes; ++i)
    {
        // determining data block amount and storage size for last reading
        if (i == readTimes - 1 && readTimes > 1)
        {
            if (fileSize % readCharacters != 0) readCharacters = fileSize % readCharacters;
            preprocess(readCharacters, &dataBlockAmountPerReading, &storageSizePerReading);
        }

        // 1. read characters from input data stream and transfer data from host to device
        C = (char *) malloc(readCharacters);
        hipMalloc((char **)&D_C, readCharacters);
        fread(C, 1, readCharacters, fin);
        hipMemcpy(D_C, C, readCharacters, hipMemcpyHostToDevice);
        free(C);

        // 2. padding characters
        CHECK(hipMalloc((unsigned char **)&D_P, storageSizePerReading));
        uint64_t blockDimension_x = 32;
        uint64_t gridDimension_x = 1;
        if (dataBlockAmountPerReading > blockDimension_x) {
            gridDimension_x = dataBlockAmountPerReading / blockDimension_x;
            if (dataBlockAmountPerReading % blockDimension_x > 0) gridDimension_x++;
        }

        printf("dataBolckAmount = %llu\n", dataBlockAmountPerReading);
        printf("blockDimension_x = %llu, gridDimension_x = %llu\n", blockDimension_x, gridDimension_x);
        getchar();
        getchar();

        dim3 block1(blockDimension_x);
        dim3 grid1(gridDimension_x);
        paddingChar<<<grid1, block1>>>((unsigned char *)D_C, D_P, DATABLOCKSIZE[0], DATABLOCKSIZE[1], PADDINGSIZE[0], PADDINGSIZE[1], dataBlockAmountPerReading);
        hipDeviceSynchronize();
        hipFree(D_C);

        getchar();
        getchar();
        
        // 3. transform 4 unsigned char to 1 32-bit unsigned int
        CHECK(hipMalloc((uint32_t **)&D_T, storageSizePerReading));
        dim3 block2(blockDimension_x);
        dim3 grid2(gridDimension_x);
        unsignedCharToUnsignedInt<<<grid2, block2>>>(D_P, D_T, DATABLOCKSIZE[0], DATABLOCKSIZE[1], PADDINGSIZE[0], PADDINGSIZE[1], dataBlockAmountPerReading);
        hipDeviceSynchronize();
        hipFree(D_P);

        // 4. extending 16 32-bit integers to 64 32-bit integers
        CHECK(hipMalloc((uint32_t **)&D_E, 4 * storageSizePerReading));
        dim3 block3(blockDimension_x);
        dim3 grid3(gridDimension_x);
        extending<<<grid3, block3>>>(D_T, D_E, DATABLOCKSIZE[0], DATABLOCKSIZE[1], PADDINGSIZE[0], PADDINGSIZE[1], dataBlockAmountPerReading);
        hipDeviceSynchronize();
        hipFree(D_T);
    
        // 5.updating hash value
        dim3 block4(blockDimension_x);
        dim3 grid4(gridDimension_x);
        updatingHashValue<<<grid4, block4>>>(D_E, D_V[0], DATABLOCKSIZE[0], DATABLOCKSIZE[1], PADDINGSIZE[0], PADDINGSIZE[1], (oddDataBlockAmount && (i == readTimes - 1)), dataBlockAmountPerReading, hashValuePosition);
        hipDeviceSynchronize();
        hipFree(D_E);
        hashValuePosition += (dataBlockAmountPerReading * 8);
    }

    // preprocess for 2 ~ (layers - 1) layer
    DATABLOCKSIZE[0] = 64LLU;
    DATABLOCKSIZE[1] = 0;
    PADDINGSIZE[0] = 64LLU;
    PADDINGSIZE[1] = 0;
    
    // computing hash value for 1 to (layers-1) layer 
    for (uint64_t l = 1; l < layers; l++) {
        // update the number of data block for per layer  
        uint64_t dataBlockAmount = hashValueAmountArray[l - 1] / 2;

        // updating storage size
        uint64_t storageSize = (DATABLOCKSIZE[0] + PADDINGSIZE[0]) * dataBlockAmount;

        // updating the parity of data block amount for per layer
        oddDataBlockAmount = false;
        if (dataBlockAmount % 2 != 0) oddDataBlockAmount = true;

        // updating the number of hash value for per layer  
        hashValueAmount = dataBlockAmount;
        if (oddDataBlockAmount && l != layers - 1) hashValueAmount++;
        hashValueAmountArray[l] = hashValueAmount;
        
        // 1. get data from the previous hash value
        hipMalloc((char **)&D_C, hashValueAmountArray[l - 1] * 8 * sizeof(uint32_t));
        hipMemcpy(D_C, D_V[l - 1], hashValueAmountArray[l - 1] * 8 * sizeof(uint32_t), hipMemcpyDeviceToDevice);

        // 2. padding characters
        CHECK(hipMalloc((char **)&D_P, storageSize));
        uint64_t blockDimension_x = 32;
        uint64_t gridDimension_x = 1;
        if (dataBlockAmount > blockDimension_x) {
            gridDimension_x = dataBlockAmount / blockDimension_x;
            if (dataBlockAmount % blockDimension_x > 0) gridDimension_x++;
        } else {
            // blockDimension_x = dataBlockAmount;
        }
        dim3 block1(blockDimension_x);
        dim3 grid1(gridDimension_x);

        printf("\n\n************layer = %llu\n", l);
        printf("dataBolckAmount = %llu\n", dataBlockAmount);
        printf("blockDimension_x = %llu, gridDimension_x = %llu\n", blockDimension_x, gridDimension_x);
        getchar();

        paddingChar<<<grid1, block1>>>((unsigned char *)D_C, D_P, DATABLOCKSIZE[0], DATABLOCKSIZE[1], PADDINGSIZE[0], PADDINGSIZE[1], dataBlockAmount);
        hipDeviceSynchronize();
        hipFree(D_C);

        getchar();

        // 3. transform 4 unsigned char to 1 32-bit unsigned int
        CHECK(hipMalloc((char **)&D_T, storageSize));
        dim3 block2(blockDimension_x);
        dim3 grid2(gridDimension_x);
        unsignedCharToUnsignedInt<<<grid2, block2>>>(D_P, D_T, DATABLOCKSIZE[0], DATABLOCKSIZE[1], PADDINGSIZE[0], PADDINGSIZE[1], dataBlockAmount);
        hipDeviceSynchronize();
        hipFree(D_P);

        // 4. extending 16 32-bit integers to 64 32-bit integers
        CHECK(hipMalloc((char **)&D_E, 4 * storageSize));
        dim3 block3(blockDimension_x);
        dim3 grid3(gridDimension_x);
        extending<<<grid3, block3>>>(D_T, D_E, DATABLOCKSIZE[0], DATABLOCKSIZE[1], PADDINGSIZE[0], PADDINGSIZE[1], dataBlockAmount);
        hipDeviceSynchronize();
        hipFree(D_T);

        // 5.updating hash value
        CHECK(hipMalloc((uint32_t **)&D_V[l], hashValueAmount * 8 * sizeof(uint32_t)));
        dim3 block4(blockDimension_x);
        dim3 grid4(gridDimension_x);
        updatingHashValue<<<grid4, block4>>>(D_E, D_V[l], DATABLOCKSIZE[0], DATABLOCKSIZE[1], PADDINGSIZE[0], PADDINGSIZE[1], oddDataBlockAmount, dataBlockAmount, 0llu);
        hipDeviceSynchronize();
        hipFree(D_E);
    }

    // assign the storage space of the hash value for per layer on host side
    uint32_t *V[layers];
    for (uint32_t i = 0; i < layers; i++) {
        V[i] = (uint32_t *)malloc(hashValueAmountArray[i] * 8 * sizeof(uint32_t));
    }

    // transfer hash value from device to host
    for (uint32_t i = 0; i < layers; i++) {
        hipMemcpy(V[i], D_V[i], hashValueAmountArray[i] * 8 * sizeof(uint32_t), hipMemcpyDeviceToHost);
    }

    // set the end time
    end = getTime();

    // present the root
    for (uint64_t j = 0; j < hashValueAmountArray[layers - 1]; j++) {
        printf("the merkle root: %08x %08x %08x %08x %08x %08x %08x %08x\n", \
                V[layers - 1][8 * j], \
                V[layers - 1][8 * j + 1], \
                V[layers - 1][8 * j + 2], \
                V[layers - 1][8 * j + 3], \
                V[layers - 1][8 * j + 4], \
                V[layers - 1][8 * j + 5], \
                V[layers - 1][8 * j + 6], \
                V[layers - 1][8 * j + 7]);
    }

    // free data pointer
    fclose(fin);
    for (uint64_t i = 0; i < layers; i++) {
        free(V[i]);
    }
    for (uint64_t i = 0; i < layers; i++) {
        hipFree(D_V[i]);
    }

    // show time consumption
    printf("time consumption: %f s\n\n", end - start);

    return 0;
}

// 0. recording time in seconds
double getTime() {
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double) tp.tv_sec + (double) tp.tv_usec * 1.e-6);
}

// 1. get data block size, padding characters, data block amount and storage size
void preprocess(const uint64_t readCharacters, uint64_t *dataBlockAmountPerReading, uint64_t *storageSizePerReading) {
    // 1. get the the size of data block for per reading
    if (readCharacters % DATABLOCKSIZE[0] > 0) {
        DATABLOCKSIZE[1] = readCharacters % DATABLOCKSIZE[0];
    }

    // 2. get the number of characters of padding for per reading
    if (DATABLOCKSIZE[0] % 64 < 56) {
        PADDINGSIZE[0] = 56 - (DATABLOCKSIZE[0] % 64) + 8;
    } else {
        PADDINGSIZE[0] = 64 - (DATABLOCKSIZE[0] % 64) + 56 + 8;
    }
    if (DATABLOCKSIZE[1] > 0) {
        if (DATABLOCKSIZE[1] % 64 < 56) {
            PADDINGSIZE[1] = 56 - (DATABLOCKSIZE[1] % 64) + 8;
        } else {
            PADDINGSIZE[1] = 64 - (DATABLOCKSIZE[1] % 64) + 56 + 8;
        }
    }

    // 3. get the number of data block for per reading
    uint64_t dataBlockAmountArray[2] = {0, 0};
    dataBlockAmountArray[0] = readCharacters / DATABLOCKSIZE[0];
    dataBlockAmountArray[1] = 0;
    if (DATABLOCKSIZE[1] > 0) dataBlockAmountArray[1] = 1;
    *dataBlockAmountPerReading = dataBlockAmountArray[0] + dataBlockAmountArray[1];
    
    // 4. get the storage size for per reading
    *storageSizePerReading = (DATABLOCKSIZE[0] + PADDINGSIZE[0]) * dataBlockAmountArray[0] + (DATABLOCKSIZE[1] + PADDINGSIZE[1]) * dataBlockAmountArray[1];
}

// 2. padding characters, data from D_C to D_P
__global__ void paddingChar(unsigned char* D_C, unsigned char* D_P, uint64_t DATABLOCKSIZE0, uint64_t DATABLOCKSIZE1, uint64_t PADDINGSIZE0, uint64_t PADDINGSIZE1, uint64_t dataBlockAmount) {
    // determining threadId
    uint64_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t iy = blockIdx.y * blockDim.y + threadIdx.y;
    uint64_t idx = iy * (gridDim.x * blockDim.x) + ix;

    if ( idx == dataBlockAmount - 1 ) {
        printf("blcokDim.x = %d\ngridDim.x = %d\n", blockDim.x, gridDim.x);
        printf("threadAmount = %d\n", (blockDim.x) * (gridDim.x));
    }

    // determining blocksize and padding size
    uint64_t dataBlockSize = DATABLOCKSIZE0;
    uint64_t paddingSize = PADDINGSIZE0; 
    if (DATABLOCKSIZE1 > 0 && idx == dataBlockAmount - 1) {
        dataBlockSize = DATABLOCKSIZE1;
        paddingSize = PADDINGSIZE1;
    }

    // initial address in D_C per thread
    uint64_t x1 = DATABLOCKSIZE0 * idx;

    // initial address in D_P per thread
    uint64_t x2 = (DATABLOCKSIZE0 + PADDINGSIZE0) * idx;

    if (idx < dataBlockAmount) {
        // cpy chars from orginal chars address to padded address  
        for (uint32_t i = 0; i < dataBlockSize; i++) {
            D_P[x2 + i] = D_C[x1 + i];
        }

        //  first time padding, padding 1000 0000
        D_P[x2 + dataBlockSize] = 0x80;

        // second time padding, padding 0000 0000, (paddingsize -9) times
        for (int i = 1; i <= paddingSize - 9; i++) {
            D_P[x2 + dataBlockSize + i] = 0x00;
        }
        
        // third time padding, padding data block length 
        for(int i = 1; i <= 8; i++) { 
            D_P[x2 + dataBlockSize + paddingSize - i] = (unsigned char)((8 * dataBlockSize) >> (i-1)*8);
        }
    }
    if (idx == 0)
    {
        printf("x2 = %llu\n", x2);
        printf("idx = %llu\n", idx);
        for (uint64_t i = 0; i < 64; i++)
        {
            printf("D_P[%llu] = %x\n", i, (uint32_t)D_P[i+x2]);
        }
    }
    if (idx == dataBlockAmount - 1)
    {
        printf("x2 = %llu\n", x2);
        printf("idx = %llu\n", idx);
        for (uint64_t i = 0; i < 64; i++)
        {
            printf("D_P[%llu] = %x\n", i, (uint32_t)D_P[i+x2]);
        }
    }
}

// 3. transform 4 unsigned char to 32-bit unsiged int
__global__ void unsignedCharToUnsignedInt(const unsigned char* D_P, uint32_t* D_T, uint64_t DATABLOCKSIZE0, uint64_t DATABLOCKSIZE1, uint64_t PADDINGSIZE0, uint64_t PADDINGSIZE1, uint64_t dataBlockAmount) {
    // determining threadId
    uint64_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t iy = blockIdx.y * blockDim.y + threadIdx.y;
    uint64_t idx = iy * (gridDim.x * blockDim.x) + ix;

    // determining blocksize and padding size
    uint64_t dataBlockSize = DATABLOCKSIZE0;
    uint64_t paddingSize = PADDINGSIZE0; 
    if (DATABLOCKSIZE1 > 0 && idx == dataBlockAmount - 1) {
        dataBlockSize = DATABLOCKSIZE1;
        paddingSize = PADDINGSIZE1;
    }

    // initial address in D_P per thread
    uint64_t x1 = (DATABLOCKSIZE0 + PADDINGSIZE0) * idx;

    // initial address in D_T per thread
    uint64_t x2 = (DATABLOCKSIZE0 + PADDINGSIZE0) / 4 * idx;

    // determining the number of groups for per data block
    uint64_t N = (dataBlockSize + paddingSize) / 64;

    if (idx < dataBlockAmount) {
        // transform
        for (uint64_t i = 0; i < N; i++) {
            D_T[x2 +  0 + 16 * i] = (D_P[x1 +  0 + 64 * i] << 24) + (D_P[x1 +  1 + 64 * i] << 16) + (D_P[x1 +  2 + 64 * i] << 8) + D_P[x1 +  3 + 64 * i];
            D_T[x2 +  1 + 16 * i] = (D_P[x1 +  4 + 64 * i] << 24) + (D_P[x1 +  5 + 64 * i] << 16) + (D_P[x1 +  6 + 64 * i] << 8) + D_P[x1 +  7 + 64 * i];
            D_T[x2 +  2 + 16 * i] = (D_P[x1 +  8 + 64 * i] << 24) + (D_P[x1 +  9 + 64 * i] << 16) + (D_P[x1 + 10 + 64 * i] << 8) + D_P[x1 + 11 + 64 * i];
            D_T[x2 +  3 + 16 * i] = (D_P[x1 + 12 + 64 * i] << 24) + (D_P[x1 + 13 + 64 * i] << 16) + (D_P[x1 + 14 + 64 * i] << 8) + D_P[x1 + 15 + 64 * i];
            D_T[x2 +  4 + 16 * i] = (D_P[x1 + 16 + 64 * i] << 24) + (D_P[x1 + 17 + 64 * i] << 16) + (D_P[x1 + 18 + 64 * i] << 8) + D_P[x1 + 19 + 64 * i];
            D_T[x2 +  5 + 16 * i] = (D_P[x1 + 20 + 64 * i] << 24) + (D_P[x1 + 21 + 64 * i] << 16) + (D_P[x1 + 22 + 64 * i] << 8) + D_P[x1 + 23 + 64 * i];
            D_T[x2 +  6 + 16 * i] = (D_P[x1 + 24 + 64 * i] << 24) + (D_P[x1 + 25 + 64 * i] << 16) + (D_P[x1 + 26 + 64 * i] << 8) + D_P[x1 + 27 + 64 * i];
            D_T[x2 +  7 + 16 * i] = (D_P[x1 + 28 + 64 * i] << 24) + (D_P[x1 + 29 + 64 * i] << 16) + (D_P[x1 + 30 + 64 * i] << 8) + D_P[x1 + 31 + 64 * i];
            D_T[x2 +  8 + 16 * i] = (D_P[x1 + 32 + 64 * i] << 24) + (D_P[x1 + 33 + 64 * i] << 16) + (D_P[x1 + 34 + 64 * i] << 8) + D_P[x1 + 35 + 64 * i];
            D_T[x2 +  9 + 16 * i] = (D_P[x1 + 36 + 64 * i] << 24) + (D_P[x1 + 37 + 64 * i] << 16) + (D_P[x1 + 38 + 64 * i] << 8) + D_P[x1 + 39 + 64 * i];
            D_T[x2 + 10 + 16 * i] = (D_P[x1 + 40 + 64 * i] << 24) + (D_P[x1 + 41 + 64 * i] << 16) + (D_P[x1 + 42 + 64 * i] << 8) + D_P[x1 + 43 + 64 * i];
            D_T[x2 + 11 + 16 * i] = (D_P[x1 + 44 + 64 * i] << 24) + (D_P[x1 + 45 + 64 * i] << 16) + (D_P[x1 + 46 + 64 * i] << 8) + D_P[x1 + 47 + 64 * i];
            D_T[x2 + 12 + 16 * i] = (D_P[x1 + 48 + 64 * i] << 24) + (D_P[x1 + 49 + 64 * i] << 16) + (D_P[x1 + 50 + 64 * i] << 8) + D_P[x1 + 51 + 64 * i];
            D_T[x2 + 13 + 16 * i] = (D_P[x1 + 52 + 64 * i] << 24) + (D_P[x1 + 53 + 64 * i] << 16) + (D_P[x1 + 54 + 64 * i] << 8) + D_P[x1 + 55 + 64 * i];
            D_T[x2 + 14 + 16 * i] = (D_P[x1 + 56 + 64 * i] << 24) + (D_P[x1 + 57 + 64 * i] << 16) + (D_P[x1 + 58 + 64 * i] << 8) + D_P[x1 + 59 + 64 * i];
            D_T[x2 + 15 + 16 * i] = (D_P[x1 + 60 + 64 * i] << 24) + (D_P[x1 + 61 + 64 * i] << 16) + (D_P[x1 + 62 + 64 * i] << 8) + D_P[x1 + 63 + 64 * i];
        }
    }
    if (idx == 0)
    {
        printf("x2 = %llu\n", x2);
        printf("idx = %llu\n", idx);
        for (uint64_t i = 0; i < 16; i++)
        {
            printf("D_T[%llu] = %x\n", i, D_T[i+x2]);
        }
    }
    if (idx == dataBlockAmount - 1)
    {
        printf("x2 = %llu\n", x2);
        printf("idx = %llu\n", idx);
        for (uint64_t i = 0; i < 16; i++)
        {
            printf("D_T[%llu] = %x\n", i, D_T[i+x2]);
        }
    }
}

// 4. extending 16 32-bit integers to 64 32-bit integers
__global__ void extending(uint32_t *D_T, uint32_t *D_E, uint64_t DATABLOCKSIZE0, uint64_t DATABLOCKSIZE1, uint64_t PADDINGSIZE0, uint64_t PADDINGSIZE1, uint64_t dataBlockAmount) {
    // determining threadId
    uint64_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t iy = blockIdx.y * blockDim.y + threadIdx.y;
    uint64_t idx = iy * (gridDim.x * blockDim.x) + ix;

    // determining blocksize and padding size
    uint64_t dataBlockSize = DATABLOCKSIZE0;
    uint64_t paddingSize = PADDINGSIZE0; 
    if (DATABLOCKSIZE1 > 0 && idx == dataBlockAmount - 1) {
        dataBlockSize = DATABLOCKSIZE1;
        paddingSize = PADDINGSIZE1;
    }

    // initial address in D_T per thread
    uint64_t x1 = (DATABLOCKSIZE0 + PADDINGSIZE0) / 4 * idx;

    // initial address in D_E per thread
    uint64_t x2 = (DATABLOCKSIZE0 + PADDINGSIZE0) * idx;

    // determining the number of groups for per data block
    uint64_t N = (dataBlockSize + paddingSize) / 64;

    if (idx < dataBlockAmount) {
        for (uint64_t i = 0; i < N; i++) {          
            D_E[x2 + 64 * i + 0] = D_T[x1 + 16 * i + 0];
            D_E[x2 + 64 * i + 1] = D_T[x1 + 16 * i + 1];
            D_E[x2 + 64 * i + 2] = D_T[x1 + 16 * i + 2];
            D_E[x2 + 64 * i + 3] = D_T[x1 + 16 * i + 3];
            D_E[x2 + 64 * i + 4] = D_T[x1 + 16 * i + 4];
            D_E[x2 + 64 * i + 5] = D_T[x1 + 16 * i + 5];
            D_E[x2 + 64 * i + 6] = D_T[x1 + 16 * i + 6];
            D_E[x2 + 64 * i + 7] = D_T[x1 + 16 * i + 7];
            D_E[x2 + 64 * i + 8] = D_T[x1 + 16 * i + 8];
            D_E[x2 + 64 * i + 9] = D_T[x1 + 16 * i + 9];
            D_E[x2 + 64 * i + 10] = D_T[x1 + 16 * i + 10];
            D_E[x2 + 64 * i + 11] = D_T[x1 + 16 * i + 11];
            D_E[x2 + 64 * i + 12] = D_T[x1 + 16 * i + 12];
            D_E[x2 + 64 * i + 13] = D_T[x1 + 16 * i + 13];
            D_E[x2 + 64 * i + 14] = D_T[x1 + 16 * i + 14];
            D_E[x2 + 64 * i + 15] = D_T[x1 + 16 * i + 15];
            for(uint64_t j=16; j < 64; j++) {
                D_E[x2 + j + 64 * i] = SSigma_1(D_E[x2 + j + 64 * i - 2]) + D_E[x2 + j + 64 * i - 7] + SSigma_0(D_E[x2 + j + 64 * i - 15]) + D_E[x2 + j + 64 * i - 16];
                D_E[x2 + j + 64 * i] = D_E[x2 + j + 64 * i] & 0xFFFFFFFF;
            }
        }
    }
    if (idx == 0)
    {
        for (uint64_t i = 0; i < 64; i++)
        {
            printf("D_E[%llu] = %x\n", i, D_E[i]);
        }
    }
    if (idx == dataBlockAmount - 1)
    {
        for (uint64_t i = 0; i < 64; i++)
        {
            printf("D_E[%llu] = %x\n", i, D_E[i]);
        }
    }
}

// 5. updating hash value
__global__ void updatingHashValue(const uint32_t *D_E, uint32_t *D_H, uint64_t DATABLOCKSIZE0, uint64_t DATABLOCKSIZE1, uint64_t PADDINGSIZE0, uint64_t PADDINGSIZE1, bool oddDataBlockAmount, uint64_t dataBlockAmount, uint64_t hashValuePosition) {    
    // determining threadId
    uint64_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t iy = blockIdx.y * blockDim.y + threadIdx.y;
    uint64_t idx = iy * (gridDim.x * blockDim.x) + ix;

    // determining blocksize and padding size
    uint64_t dataBlockSize = DATABLOCKSIZE0;
    uint64_t paddingSize = PADDINGSIZE0; 
    if (DATABLOCKSIZE1 > 0 && idx == dataBlockAmount - 1) {
        dataBlockSize = DATABLOCKSIZE1;
        paddingSize = PADDINGSIZE1;
    }

    // initial address in D_E per thread
    uint64_t x1 = (DATABLOCKSIZE0 + PADDINGSIZE0) * idx;

    // initial address in D_H per thread
    uint64_t x2 = 8 * idx;

    // determining the number of groups for per data block
    uint64_t N = (dataBlockSize + paddingSize) / 64;

    // preprocess
    uint32_t t1, t2, h1, h2, h3, h4, h5, h6, h7, h8;

    D_H[x2 + 0 + hashValuePosition] = h1 = 0x6a09e667;
    D_H[x2 + 1 + hashValuePosition] = h2 = 0xbb67ae85;
    D_H[x2 + 2 + hashValuePosition] = h3 = 0x3c6ef372;
    D_H[x2 + 3 + hashValuePosition] = h4 = 0xa54ff53a;
    D_H[x2 + 4 + hashValuePosition] = h5 = 0x510e527f;
    D_H[x2 + 5 + hashValuePosition] = h6 = 0x9b05688c;
    D_H[x2 + 6 + hashValuePosition] = h7 = 0x1f83d9ab;
    D_H[x2 + 7 + hashValuePosition] = h8 = 0x5be0cd19;

    const uint32_t K[64] = {
            0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
            0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
            0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
            0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
            0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
            0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
            0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
            0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2,
    };

    // cycliclly updating hash value
    if (idx < dataBlockAmount) {
        for (uint32_t i = 0; i < N; i++) {
            for (uint32_t j = 0; j < 64; j++) {
                t1 = (h8 + LSigma_1(h5) + Conditional(h5, h6, h7) + K[j] + D_E[x1 + j + 64 * i]) & 0xFFFFFFFF;
                t2 = (LSigma_0(h1) + Majority(h1, h2, h3)) & 0xFFFFFFFF;
                h8 = h7;
                h7 = h6;
                h6 = h5;
                h5 = (h4 + t1) & 0xFFFFFFFF;
                h4 = h3;
                h3 = h2;
                h2 = h1;
                h1 = (t1 + t2) & 0xFFFFFFFF;
            }
            D_H[x2 + 0 + hashValuePosition] = (D_H[x2 + 0 + hashValuePosition] + h1) & 0xFFFFFFFF;
            D_H[x2 + 1 + hashValuePosition] = (D_H[x2 + 1 + hashValuePosition] + h2) & 0xFFFFFFFF;
            D_H[x2 + 2 + hashValuePosition] = (D_H[x2 + 2 + hashValuePosition] + h3) & 0xFFFFFFFF;
            D_H[x2 + 3 + hashValuePosition] = (D_H[x2 + 3 + hashValuePosition] + h4) & 0xFFFFFFFF;
            D_H[x2 + 4 + hashValuePosition] = (D_H[x2 + 4 + hashValuePosition] + h5) & 0xFFFFFFFF;
            D_H[x2 + 5 + hashValuePosition] = (D_H[x2 + 5 + hashValuePosition] + h6) & 0xFFFFFFFF;
            D_H[x2 + 6 + hashValuePosition] = (D_H[x2 + 6 + hashValuePosition] + h7) & 0xFFFFFFFF;
            D_H[x2 + 7 + hashValuePosition] = (D_H[x2 + 7 + hashValuePosition] + h8) & 0xFFFFFFFF;
            h1 = D_H[x2 + 0 + hashValuePosition];
            h2 = D_H[x2 + 1 + hashValuePosition];
            h3 = D_H[x2 + 2 + hashValuePosition];
            h4 = D_H[x2 + 3 + hashValuePosition];
            h5 = D_H[x2 + 4 + hashValuePosition];
            h6 = D_H[x2 + 5 + hashValuePosition];
            h7 = D_H[x2 + 6 + hashValuePosition];
            h8 = D_H[x2 + 7 + hashValuePosition];
        }
    }

    if (idx == 0)
    {
        for (uint64_t i = 0; i < 8; i++)
        {
            printf("idx = %llu, D_H[%llu] = %x\n", idx, i, D_H[i + hashValuePosition]);
        }
        
    }
    if (idx == dataBlockAmount - 1)
    {
        for (uint64_t i = 0; i < 8; i++)
        {
            printf("idx = %llu, D_H[%llu] = %x\n", idx, i, D_H[i + hashValuePosition]);
        }
        
    }
    
    // when the number of hash vaule amount is odd, copy the last-1 hash value
    if (oddDataBlockAmount && (idx == dataBlockAmount - 1)) {
        D_H[8 * dataBlockAmount + 0 + hashValuePosition] = D_H[8 * dataBlockAmount - 8 + hashValuePosition];
        D_H[8 * dataBlockAmount + 1 + hashValuePosition] = D_H[8 * dataBlockAmount - 7 + hashValuePosition];
        D_H[8 * dataBlockAmount + 2 + hashValuePosition] = D_H[8 * dataBlockAmount - 6 + hashValuePosition];
        D_H[8 * dataBlockAmount + 3 + hashValuePosition] = D_H[8 * dataBlockAmount - 5 + hashValuePosition];
        D_H[8 * dataBlockAmount + 4 + hashValuePosition] = D_H[8 * dataBlockAmount - 4 + hashValuePosition];
        D_H[8 * dataBlockAmount + 5 + hashValuePosition] = D_H[8 * dataBlockAmount - 3 + hashValuePosition];
        D_H[8 * dataBlockAmount + 6 + hashValuePosition] = D_H[8 * dataBlockAmount - 2 + hashValuePosition];
        D_H[8 * dataBlockAmount + 7 + hashValuePosition] = D_H[8 * dataBlockAmount - 1 + hashValuePosition];
    }
}